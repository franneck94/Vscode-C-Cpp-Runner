#include "hip/hip_runtime.h"


#include <iostream>
#include <stdlib.h>

#define N 10000000

__global__ void vector_add(float *out, float *a, float *b, int n)
{
    for (int i = 0; i < n; i++)
    {
        out[i] = a[i] + b[i];
    }
}

int main()
{
    float *a = NULL;
    float *b = NULL;
    float *d_a = NULL;
    float *d_b = NULL;
    float *out = NULL;

    a = (float *)malloc(sizeof(float) * N);
    b = (float *)malloc(sizeof(float) * N);

    // Allocate device memory for a
    hipMalloc((void **)&d_a, sizeof(float) * N);
    hipMalloc((void **)&d_b, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    vector_add<<<1, 1>>>(out, d_a, d_b, N);

    // Cleanup after kernel execution
    hipFree(d_a);
    hipFree(d_b);
    free(a);
    free(b);

    std::cout << "Test\n";

    return 0;
}
